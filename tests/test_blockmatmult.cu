#include "hip/hip_runtime.h"
/**
	Let A be a Laplace (tridiagonal) matrix

    We are interested in the comparison of several ways how to compute A*x with MINLIN:
    TEST_MINLIN_FULL - create dense minlin-matrix and multiply with it
    TEST_MINLIN      - use idea from Ben: Ax = -x(..) + 2*x(..) - x(..)
    TEST_FOR         - use naive sequential "for" cycle
    TEST_OMP         - run the previous "for" cycle as OpenMP "parallel for"
    TEST_CUDA        - iteration of "for" cycle is runned as CUDA kernel

**/

#include <minlin/minlin.h>
#include <minlin/modules/threx/threx.h>

#include <iostream>

#include <stdio.h> /* printf in cuda */
#include <limits> /* max value of double/float */

#include <omp.h>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <>
#include <hip/device_functions.h>

using namespace minlin::threx;

/* compute on device or host ? and which tests to run ? */
#ifdef USE_GPU
	/* compute using CUDA on Device */

	#define MyVector DeviceVector
	#define MyMatrix DeviceMatrix

	#define TEST_MINLIN_FULL false
	#define TEST_MINLIN true
	#define TEST_FOR false
	#define TEST_OMP false
	#define TEST_CUDA true

#else
	/* compute without CUDA on Host */

	#define MyVector HostVector
	#define MyMatrix HostMatrix

	#define TEST_MINLIN_FULL false
	#define TEST_MINLIN true
	#define TEST_FOR true
	#define TEST_OMP true
	#define TEST_CUDA false

#endif

/* double/float values in Vector? */
#define Scalar double

MINLIN_INIT

/* cuda error check */
#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess) 
   {
      fprintf(stderr,"\n\x1B[31mCUDA error:\x1B[0m %s %s \x1B[33m%d\x1B[0m\n\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}

/* timer */
double getUnixTime(void){
	struct timespec tv;
	if(clock_gettime(CLOCK_REALTIME, &tv) != 0) return 0;
	return (((double) tv.tv_sec) + (double) (tv.tv_nsec / 1000000000.0));
}


/* A*x using MINLIN matrix-vector multiplication (with dense matrix) */
void my_multiplication_minlin_full(MyVector<Scalar> *Ax, MyMatrix<Scalar> A, MyVector<Scalar> x){
	(*Ax) = A*x;
}

/* A*x using MINLIN with vectors (idea from Ben) */
void my_multiplication_minlin(MyVector<Scalar> *Ax, MyVector<Scalar> x, int K){
	int N = x.size();
	int T = N/K;
	int k;
	
	(*Ax)(1,N-2) = 2*x(1,N-2) - x(0,N-3) - x(2,N-1);
	
	/* first and last in each block */
	for(k=0;k<K;k++){
		(*Ax)(k*T) = x(k*T) - x(k*T+1);
		(*Ax)((k+1)*T-1) = x((k+1)*T-1) - x((k+1)*T-2);
	}
	
	
}


/* A*x using simple sequential "for" cycle */
void my_multiplication_for(MyVector<Scalar> *Ax, MyVector<Scalar> x, int K){
	int N = x.size();
	int T = N/K;
	int t,k, id_row;

	for(k=0;k<K;k++){
		for(t=0;t<T;t++){
			id_row = k*T+t;

			/* first row */
			if(t == 0){
				(*Ax)(id_row) = x(id_row) - x(id_row+1);
			}
			/* common row */
			if(t > 0 && t < T-1){
				(*Ax)(id_row) = -x(id_row-1) + 2.0*x(id_row) - x(id_row+1);
			}
			/* last row */
			if(t == T-1){
				(*Ax)(id_row) = -x(id_row-1) + x(id_row);
			}
		}
	}
}

/* A*x using OpenMP */
void my_multiplication_omp(MyVector<Scalar> *Ax, MyVector<Scalar> x, int K){
	int N = x.size();
	int T = N/K;
	int t,k, id_row;

	#pragma omp parallel for private(k)
	for(k=0;k<K;k++){

		#pragma omp parallel for private(t)
		for(t=0;t<T;t++){
			id_row = k*T+t;

			/* first row */
			if(t == 0){
				(*Ax)(id_row) = x(id_row) - x(id_row+1);
			}
			/* common row */
			if(t > 0 && t < T-1){
				(*Ax)(id_row) = -x(id_row-1) + 2.0*x(id_row) - x(id_row+1);
			}
			/* last row */
			if(t == T-1){
				(*Ax)(id_row) = -x(id_row-1) + x(id_row);
			}
		}
	}
	
}

/* A*x using CUDA kernel */
template <typename T> __global__
void kernel_mult(T* Axp, T* xp, int T, int K)
{
	/* compute my id */
	int t = blockIdx.x*blockDim.x + threadIdx.x;

	/* compute id of cluster */
	int k = (int)(t/T);
	
	/* compute id_row in local block */
	int t_local = t-k*T;

	/* test access to array with vector values */
//	printf("x(%d) = %f\n",t,xp[t]);

	/* first row */
	if(t_local == 0){
		Axp[t] = xp[t] - xp[t+1];
	}
	/* common row */
	if(t_local > 0 && t_local < T-1){
		Axp[t] = -xp[t-1] + 2.0*xp[t] - xp[t+1];
	}
	/* last row */
	if(t_local == T-1){
		Axp[t] = -xp[t-1] + xp[t];
	}

	/* if t >= N then relax and do nothing */	

}

void my_multiplication_cuda(MyVector<Scalar> *Ax, MyVector<Scalar> x, int K){
	int N = x.size();
	int T = N/K;

	/* call cuda kernels */
	/* pass a thrust raw pointers to cuda kernel */
	Scalar *xp = x.pointer(); /* thank minlin for this function! */
	Scalar *Axp = (*Ax).pointer();

	kernel_mult<<<N, 1>>>(Axp,xp,T,K);

	/* synchronize kernels, if there is an error with cuda, then it will appear here */ 
	gpuErrchk( hipDeviceSynchronize() );
}

/* fill vector using CUDA kernel */
template <typename T> __global__
void fill_x(T* x, int N)
{
	/* compute my id */
	int t = blockIdx.x*blockDim.x + threadIdx.x;

	if(t < N){
		x[t] = 1.0 + 1.0/(Scalar)(t+1);;
	}
	
	/* if t >= N then relax and do nothing */	
}



int main ( int argc, char *argv[] ) {

	/* read command line arguments */
	if(argc < 3){
		std::cout << "1. argument - T - the dimension of the subproblem" << std::endl;
		std::cout << "2. argument - K - number of clusters" << std::endl;
		std::cout << "3. argument - M - number of tests (default 10)" << std::endl;

		std::cout << std::endl << argv[0] << " T K" << std::endl;
		std::cout << argv[0] << " T K M" << std::endl;
		return 1;
	}

	int k; /* iterators */
	int T = atoi(argv[1]); /* the first argument is the dimension of subproblem */
	std::cout << "T = " << T << " (dimension of subproblem)" << std::endl;
	int K = atoi(argv[2]); /* the second argument is the number of clusters */
	std::cout << "K = " << K << " (number of clusters)" << std::endl;

	int M = 10; /* default number of tests */
	if(argc >= 4){
		M = atoi(argv[3]); /* the third (optional) argument is the number of tests */
	}
	std::cout << "M = " << M << " (number of tests)" << std::endl;

	int N = T*K; /* dimension of problem */
	std::cout << "N = " << N << " (dimension of problem)" << std::endl;

	double t_start, t_end; /* to measure time */

	/* fill vector with some values */
	t_start = getUnixTime();
	MyVector<Scalar> x(N);
	x(all) = 0.0;
	#ifdef USE_GPU
		/* fill vector using CUDA */
		// TODO: optimal number of threads/block
		Scalar *xp = x.pointer();
		fill_x<<<N, 1>>>(xp,N);
		gpuErrchk( hipDeviceSynchronize() );
		
	#else
		/* fill vector using OpenMP */
		#pragma omp parallel for private(k)
		for(k=0;k<N;k++){
			/* vector */
			x(k) = 1.0 + 1.0/(Scalar)(k+1);
		}	
		
	#endif
		
	std::cout << "init & fill vector: " << getUnixTime() - t_start << "s" << std::endl;


	/* if it is MINLIN_FULL test, create&fill the matrix */
	#if TEST_MINLIN_FULL
		t_start = getUnixTime();

		MyMatrix<Scalar> A(N,N);

		A(all) = 0.0;

		int t,id_row;
		for(k=0;k<K;k++){
			for(t=0;t<T;t++){
				id_row = k*T + t;
				/* first row */
				if(t == 0){
					A(id_row,id_row) = 1.0;
					A(id_row,id_row+1) = -1.0;
				}
				/* common row */
				if(t > 0 && t < T-1){
					A(id_row,id_row+1) = -1.0;
					A(id_row,id_row) = 2.0;
					A(id_row,id_row-1) = -1.0;
				}
				/* last row */
				if(t == T-1){
					A(id_row,id_row-1) = -1.0;
					A(id_row,id_row) = 1.0;
				}
			}
		}	
		std::cout << "init & fill matrix: " << getUnixTime() - t_start << "s" << std::endl;

		if(N <= 15) std::cout << "A:" << A << std::endl;

	#endif

	std::cout << std::endl;

	/* to compute average time of each algorithm */
	/* these variables store the sum of computing times */
	#if TEST_MINLIN_FULL
		double t_minlin_full = 0.0;
	#endif
	#if TEST_MINLIN
		double t_minlin = 0.0;
	#endif
	#if TEST_FOR
		double t_for = 0.0;
	#endif
	#if TEST_OMP
		double t_omp = 0.0;
	#endif
	#if TEST_CUDA
		double t_cuda = 0.0;
	#endif

	/* multiplication test */
	MyVector<Scalar> Ax(N);
	
	/* I want to see the problems with setting the vector values immediately in the norm */
	/* if I forget to set a component of Ax, then the norm will be huge */
	Scalar default_value = std::numeric_limits<Scalar>::max(); 

	for(k = 0;k < M;k++){ /* for every test */
		std::cout << k+1 << ". test (of " << M << ")" << std::endl;
		
		#if TEST_MINLIN_FULL
			Ax(all) = default_value; /* clean previous results */

			t_start = getUnixTime();
			my_multiplication_minlin_full(&Ax, A, x);
			t_end = getUnixTime() - t_start;

			std::cout << " minlin_full: " << t_end << "s, norm(Ax) = " << norm(Ax) << std::endl;
			t_minlin_full += t_end;

			/* if the dimension is small, then show also the content */
			if(N <= 15) std::cout << "  " << Ax << std::endl;	
		#endif

		#if TEST_MINLIN
			Ax(all) = default_value; /* clean previous results */

			t_start = getUnixTime();
			my_multiplication_minlin(&Ax, x, K);
			t_end = getUnixTime() - t_start;

			std::cout << " minlin: " << t_end << "s, norm(Ax) = " << norm(Ax) << std::endl;
			t_minlin += t_end;

			if(N <= 15) std::cout << "  " << Ax << std::endl;	
		#endif
		
		#if TEST_FOR
			Ax(all) = default_value; /* clean previous results */

			t_start = getUnixTime();
			my_multiplication_for(&Ax, x, K);
			t_end = getUnixTime() - t_start;

			std::cout << " for:    " << t_end << "s, norm(Ax) = " << norm(Ax) << std::endl;
			t_for += t_end;

			if(N <= 15) std::cout << "  " << Ax << std::endl;	
		#endif
		
		#if TEST_OMP
			Ax(all) = default_value; /* clean previous results */

			t_start = getUnixTime();
			my_multiplication_omp(&Ax, x, K);
			t_end = getUnixTime() - t_start;

			std::cout << " omp:    " << t_end << "s, norm(Ax) = " << norm(Ax) << std::endl;
			t_omp += t_end;

			if (N <= 15) std::cout << "  " << Ax << std::endl;

		#endif

		#if TEST_CUDA
			Ax(all) = default_value; /* clean previous results */

			t_start = getUnixTime();
			my_multiplication_cuda(&Ax, x, K);
			t_end = getUnixTime() - t_start;

			std::cout << " cuda:   " << t_end << "s, norm(Ax) = " << norm(Ax) << std::endl;
			t_cuda += t_end;

			if(N <= 15) std::cout << "  " << Ax << std::endl;	
		#endif

		std::cout << "-----------------------------------------------------------" << std::endl;

	}
	
	
	
	/* give final info with average times */
	std::cout << std::endl;
	std::cout << "T = " << T << " (dimension of subproblem)" << std::endl;
	std::cout << "K = " << K << " (number of clusters)" << std::endl;
	std::cout << "N = " << N << " (dimension of problem)" << std::endl;
	std::cout << "M = " << M << " (number of tests)" << std::endl;
	std::cout << "average times:" << std::endl;

	/* compute and show average computing times */
	#if TEST_MINLIN_FULL
		std::cout << "minlin_full: " << t_minlin_full/(double)M << std::endl;
	#endif
	#if TEST_MINLIN
		std::cout << "minlin:      " << t_minlin/(double)M << std::endl;
	#endif
	#if TEST_FOR
		std::cout << "for:         " << t_for/(double)M << std::endl;
	#endif
	#if TEST_OMP
		std::cout << "omp:         " << t_omp/(double)M << std::endl;
	#endif
	#if TEST_CUDA
		std::cout << "cuda:        " << t_cuda/(double)M << std::endl;
	#endif

	
}
