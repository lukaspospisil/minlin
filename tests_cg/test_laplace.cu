#include "hip/hip_runtime.h"

/* cout */
#include <iostream>

/* max value of double/float */
#include <limits> 

/* openmpi */
#include <omp.h>

/* minlin */
#include <minlin/minlin.h>
#include <minlin/modules/threx/threx.h>

/* petsc */
#include "petsc.h"

/* my petsc stuff */
#include "petscvector.h"
#include "laplacefullmatrix.h"
#include "cg.h"


using namespace minlin::threx;
MINLIN_INIT


int DEBUG_MODE = 0;
bool PETSC_INITIALIZED = false;

//typedef HostVector<double> HostVectorDouble;

/* timer */
double getUnixTime(void){
	struct timespec tv;
	if(clock_gettime(CLOCK_REALTIME, &tv) != 0) return 0;
	return (((double) tv.tv_sec) + (double) (tv.tv_nsec / 1000000000.0));
}



int main ( int argc, char *argv[] ) {

	/* read command line arguments */
	if(argc < 2){
		std::cout << "1. argument - N - the dimension of the problem" << std::endl;

		std::cout << std::endl << argv[0] << " N" << std::endl;
		return 1;
	}

	int N = atoi(argv[1]); /* the first argument is the dimension of problem */
	std::cout << "N = " << N << " (dimension)" << std::endl;

	double t_start, t; /* to measure time */


	/* -------------------------------- PETSC TEST -------------------------*/

	std::cout << "-------------------------------- PETSC TEST -------------------------" << std::endl;

	/* initialize Petsc */
	PetscInitialize(&argc,&argv,PETSC_NULL,PETSC_NULL);
	PETSC_INITIALIZED = true;
	
	/* create vectors */
	PetscVector x0(N); /* create basic layout */
	x0(all) = 0.0;

	PetscVector b(x0); /* initialize, duplicate layout of x0 */
	b(all) = 1.0;
	b(0) = 0.0;
	b(N-1) = 0.0;

	LaplaceFullMatrix<PetscVector> A(x0); /* prepare laplace matrix */

	PetscVector x; /* solution */

	/* do some fun */
	t_start = getUnixTime();

	x = cg(A, b, x0);

	t = getUnixTime() - t_start;

	/* sufficiently small problem - give info */
	if(N <= 10){
		std::cout << "A: " << A << std::endl;
		std::cout << "x0: " << x0 << std::endl;
		std::cout << "b: " << b << std::endl;
		std::cout << "x: " << x << std::endl;
	}
	std::cout << "time: " << t << "s" << std::endl;

	PETSC_INITIALIZED = false;
	PetscFinalize();


	/* -------------------------------- MINLINHOST TEST -------------------------*/

	std::cout << "-------------------------------- MINLINHOST TEST -------------------------" << std::endl;

	/* create vectors */
	HostVectorD x0m(N);
	x0m(all) = 0.0;

	HostVectorD bm(N);
	bm(all) = 1.0;
	bm(0) = 0.0;
	bm(N-1) = 0.0;

	LaplaceFullMatrix<HostVectorD> Am(x0m); /* prepare laplace matrix */

	HostVectorD xm(N); /* solution */

	/* do some fun */
	t_start = getUnixTime();

	xm = cg(Am, bm, x0m);

	t = getUnixTime() - t_start;

	/* sufficiently small problem - give info */
	if(N <= 10){
		std::cout << "A: " << Am << std::endl;
		std::cout << "x0: " << x0m << std::endl;
		std::cout << "b: " << bm << std::endl;
		std::cout << "x: " << xm << std::endl;
	}
	std::cout << "time: " << t << "s" << std::endl;

	
}
